#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define DEBUG

// nvcc -o dot_prod dot_prod.cu
// ./dot_prod 31 28 6 2b


static int n;
static float dot_parallel;
static float dot_serial;
#define THREADS_PER_BLOCK = 32 //only use at task 3


float Serial_dot_prod(float x[], float y[], int n) {//serial
    float cdot = 0.0;

    for (int i = 0; i < n; i++) {
        cdot += x[i] * y[i];
    }
    return cdot;
}

//GPU Kernel
__global__ void dot_1(float *a, float *b, float *c, int n)//basic parallel, assume thred num = vector size
{
    //Assuming n is the number of threads, using atomicAdd


    //compute each threads' corresponding array index.
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    //then, use atomicAdd to add the result to c; you can assume each thread will be responsible for only one element
    if (index < n) {
        atomicAdd(c, a[index] * b[index]);

    }

}

//GPU Kernel
__global__ void dot_2a(float *a, float *b, float *c, int n)//block distribution
{
    // 2a: block distribution

    // compute array elems per block
    int num_per_block, num_per_thread, start_index, end_index;
    if (n < gridDim.x) {
        num_per_block = 1;
    } else {
        num_per_block = n / gridDim.x;
    }

    // compute array elems per thread
    if (num_per_block < blockDim.x) {
        num_per_thread = 1;
    } else {
        num_per_thread = num_per_block / blockDim.x;
    }

    if (num_per_block == 1) {

    }
    start_index = num_per_thread * blockDim.x * blockIdx.x + num_per_thread * threadIdx.x;

    if (start_index >= n) {
        end_index = start_index;
    } else if ((blockIdx.x == (gridDim.x - 1)) && (threadIdx.x == (blockDim.x - 1))) {
        end_index = n;
    } else {
        end_index = num_per_thread * blockDim.x * blockIdx.x + num_per_thread * (threadIdx.x + 1);
    }

    float current = 0.0;
    for (int i = start_index; i < end_index; i++) {
        if (i >= n) {
            break;
        }
        current += a[i] * b[i];
    }
    atomicAdd(c, current);

    // loop through each thread's responsible task; achieve block distribution WITHIN each block.
    //Work for multiple blocks and when thread number != array length.


}

__global__ void dot_2b(float *a, float *b, float *c, int n)//cyclic distribution
{
    // 2b: cyclic distribution

    // compute array elems per block
    int total_num_thread = gridDim.x * blockDim.x;

    int num_per_block, num_per_thread;
    if (n < gridDim.x) {
        num_per_block = 1;
    } else {
        num_per_block = n / gridDim.x; //round down
    }

    // compute array elems per thread
    if (num_per_block < blockDim.x) {
        num_per_thread = 1;
    } else {
        num_per_thread = num_per_block / blockDim.x; //round down
    }

    int index = blockDim.x * blockIdx.x + threadIdx.x;


    float current = 0.0;
    //Use <= to have one more iteration, since previous steps always round down
    for (int i = 0; i <= num_per_thread; i++) {
        if (index >= n) {
            break;
        }
        current += a[index] * b[index];
        index += total_num_thread;

    }
    atomicAdd(c, current);

    // loop through each thread's responsible task; achieve cyclic distribution WITHIN each block.
    //Work for multiple blocks and when thread number != array length.

}


//GPU Kernel
__global__ void dot_3(float *a, float *b, float *c, int n)//shared memory
{
    //3: optimize with shared memory

    // allocate a shared array; they will be shared within block; assume threads per block is 32;
    __shared__ float my_shared_array[32];
    my_shared_array[threadIdx.x] = 0.0;

    __syncthreads();

    int total_num_thread = gridDim.x * blockDim.x;
    // compute array elems per block
    int num_per_block, num_per_thread;
    if (n < gridDim.x) {
        num_per_block = 1;
    } else {
        num_per_block = n / gridDim.x; //round down
    }

    // compute array elems per thread
    if (num_per_block < blockDim.x) {
        num_per_thread = 1;
    } else {
        num_per_thread = num_per_block / blockDim.x; //round down
    }

    int index = blockDim.x * blockIdx.x + threadIdx.x;


    //Use <= to have one more iteration, since previous steps always round down
    for (int i = 0; i <= num_per_thread; i++) {
        if (index >= n) {
            break;
        }
        my_shared_array[threadIdx.x] += a[index] * b[index];
        index += total_num_thread;

    }//loop through each thread's responsible task, utilize shared memory.
    __syncthreads();
    float current = 0.0;
    for (int i = 0; i < 32; i++) {
        current += my_shared_array[i];

        if ((threadIdx.x == 31) && (i == 31)) {
            atomicAdd(c, current);
        }
    }
   //one thread per block add the partial sum saved in shared memory to the result c.
    //Find a way to ensure every thread finishes before adding.

}


void Init_vectors(float x[], float y[], int n) {
    for (int i = 0; i < n; i++) {
        // Generate a random number
        float x_val = (float) rand() / RAND_MAX;
        float y_val = (float) rand() / RAND_MAX;
        // Put the random number into the correct array cell
        x[i] = x_val;
        y[i] = y_val;
    }
#ifdef DEBUG
    printf("X values:\n");
    for (int i = 0; i < n; i++) {
        printf("%f\n", x[i]);
    }
    printf("Y values:\n");
    for (int i = 0; i < n; i++) {
        printf("%f\n", y[i]);
    }
#endif
}

void Allocate_vectors(float **x_p, float **y_p, float **dot_p, int n) {
    hipMallocManaged(x_p, n * sizeof(float));
    hipMallocManaged(y_p, n * sizeof(float));
    hipMallocManaged(dot_p, sizeof(float));

}

void Free_vectors(float *x, float *y, float *dot) {
    hipFree(x);
    hipFree(y);
    hipFree(dot);
}


int main(int argc, char *argv[]) {
    int th_per_blk, blk_ct;
    if (argc != 5) {
        printf(" Number of params wrong\n");
        return -1;
    }
    n = atoi(argv[1]);
    blk_ct = atoi(argv[2]);
    th_per_blk = atoi(argv[3]);
    char *kernel = argv[4]; //kernel name: 1,2a,2b,3
    printf("%s\n", kernel);

    //declare variables cpu/device
    float x_cpu[n], y_cpu[n]; //n is static by cuda's need
    float *x_device, *y_device, *dot_device;
    Init_vectors(x_cpu, y_cpu, n);//random fill vectors (on cpu)

    //time serial computation
    double start_serial, finish_serial, elapsed_serial;
    double start, finish, elapsed;
    GET_TIME(start_serial);
    dot_serial = Serial_dot_prod(x_cpu, y_cpu, n);//serial dot product
    GET_TIME(finish_serial);
    elapsed_serial = finish_serial - start_serial;
    printf("Serial computation took %e seconds\n", elapsed_serial);

    //alloc cuda memory
    Allocate_vectors(&x_device, &y_device, &dot_device, n);

    //copy array to and from GPU
    hipMemcpy(x_device, x_cpu, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_device, y_cpu, n * sizeof(float), hipMemcpyHostToDevice);

    GET_TIME(start);
    //call kernel
    if (!strcmp(kernel, "1"))
        dot_1<<< blk_ct, th_per_blk>>>(x_device, y_device, dot_device, n);
    else if (!strcmp(kernel, "2a"))
        dot_2a<<< blk_ct, th_per_blk>>>(x_device, y_device, dot_device, n);
    else if (!strcmp(kernel, "2b"))
        dot_2b<<< blk_ct, th_per_blk>>>(x_device, y_device, dot_device, n);
    else if (!strcmp(kernel, "3"))
        dot_3<<< blk_ct, th_per_blk>>>(x_device, y_device, dot_device, n);
    else {
        printf("Kernel name not right \n");
        return -1;
    }
    hipDeviceSynchronize();
    GET_TIME(finish);
    hipMemcpy(&dot_parallel, dot_device, sizeof(float), hipMemcpyDeviceToHost);
    elapsed = finish - start;
    printf("Parallel computation took %e seconds\n", elapsed);

    //save parallel result to `dot_parallel`

    if (fabs(dot_parallel - dot_serial) < 1e-3 * th_per_blk * blk_ct)
        printf("Result is CORRECT; parallel result: %f, serial result: %f \n", dot_parallel, dot_serial);
    else
        printf("Result is FALSE; parallel result: %f, serial result: %f \n", dot_parallel, dot_serial);

    //free cuda vectors
    Free_vectors(x_device, y_device, dot_device);

}
